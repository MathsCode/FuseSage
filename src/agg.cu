#include "../include/agg.h"

void update(float *feature,
            float *weights, 
            float *update_results,
            int M, 
            int K, 
            int N, 
            hipStream_t stream)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);
    float alpha = 1.f, beta = 0.f;
    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                N,
                M,
                K,
                &alpha,
                weights,
                N,
                feature,
                K,
                &beta,
                update_results,
                N);
}

void upward(float *update_results, int nnz,
            int32_t *csr_indptr_d, int32_t *csr_indice_d, float *csr_value_d,
            float *upward_results, int M, int K, int N, hipStream_t stream)
{
    // Run cuSparseSpMM
    // Warning: upward need add bias
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, stream);
    hipsparseDnMatDescr_t UpdateDescr, UpwardDescr;
    hipsparseSpMatDescr_t Adj;

    hipsparseCreateDnMat(&UpdateDescr, K, N, N, update_results, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseCreateDnMat(&UpwardDescr, M, N, N, upward_results, HIP_R_32F, HIPSPARSE_ORDER_ROW);


    hipsparseCreateCsr(&Adj, M, K, nnz, csr_indptr_d, csr_indice_d, csr_value_d,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO,
                      HIP_R_32F);

    size_t workspace_size = 0;
    float alpha = 1.0f, beta = 0.0f;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, Adj, UpdateDescr, &beta,
                            UpwardDescr, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT,
                            &workspace_size);
    void *workspace = NULL;
    hipMalloc(&workspace, workspace_size);
    hipsparseSpMM(handle,
                 HIPSPARSE_OPERATION_NON_TRANSPOSE, // opA
                 HIPSPARSE_OPERATION_NON_TRANSPOSE, // opB
                 &alpha, Adj, UpdateDescr, &beta, UpwardDescr,
                 HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, workspace);
}